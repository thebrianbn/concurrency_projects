// Author: Brian Nguyen


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 1000 // number of array elements
#define B 1024  // number of elements in a block

__global__ void scan(float *g_odata, float *g_idata, int n);
__global__ void prescan(float *g_odata, float *g_idata, int n, float *g_sums);
void scanCPU(float *f_out, float *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end) {
	/* Calculate the time difference. */

	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
} 

int main() {
	/* Compare results between serial and parallel versions of the
	prefix-sums algorithm. */

	int grid_size = floor(N / B) + 1;
	int thread_size = B / 2;

	// arrays to be used for initial, cpu-result, and gpu-result arrays
	// respectively.
	float a[N], c[N], g[N], sums[grid_size], inc[grid_size], sums_inc[grid_size];
	timeval start, end;

	// temporary pointer arrays for computation
	float *dev_a, *dev_g, *dev_sums, *dev_inc, *dev_sums_inc;
	int size = N * sizeof(float);
	int size_sums = grid_size * sizeof(float);

	double d_gpuTime, d_cpuTime;

	// initialize matrix a with random floats between 0 and 1000
	for (int i = 0; i < N; i++) {
		a[i] = (float)(rand() % 1000000) / 1000.0;
	}

	// initialize a and b matrices here for CUDA
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);
	hipMalloc((void **) &dev_sums, size_sums);

	// GPU version (CUDA) of prefix-sum
	gettimeofday(&start, NULL);
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	//scan<<<1,N,2*N*sizeof(float)>>>(dev_g, dev_a, N);  // naive scan
	prescan<<<grid_size, thread_size, B*sizeof(float)>>>(dev_g, dev_a, N, dev_sums);  // work-efficient scan
	hipDeviceSynchronize();
	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(sums, dev_sums, size_sums, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	d_gpuTime = myDiffTime(start, end);

	// CPU version (serial) of prefix-sum
	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);
	
	hipFree(dev_a); hipFree(dev_g);

	// display results of the prefix-sum
	for (int i = 0; i < N; i++) {
		printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
		//if (c[i] != g[i])
		//{
		//	printf("Results do not match! c[%i]=%f, g[%i]=%f\n", i, c[i], i, g[i]);
		//	break;
		//}
	}

	for (int j = 0; j < grid_size; j++) {
		printf("sums[%i] = %0.3f\n", j, sums[j]);
	}

	hipMalloc((void **) &dev_sums, size_sums);
	hipMalloc((void **) &dev_inc, size_sums);
	hipMalloc((void **) &dev_sums_inc, size_sums);

	hipMemcpy(dev_sums, sums, size_sums, hipMemcpyHostToDevice);

	prescan<<<grid_size/4, thread_size, B*sizeof(float)>>>(dev_inc, dev_sums, size_sums, dev_sums_inc);

	hipMemcpy(inc, dev_inc, size_sums, hipMemcpyDeviceToHost);
	hipMemcpy(sums_inc, dev_sums_inc, size_sums, hipMemcpyDeviceToHost);

	for (int j = 0; j < grid_size; j++) {
		printf("inc[%i] = %0.3f\n", j, inc[j]);
	}

	for (int i = 0; i < grid_size; i++) {
		for (int j = 0; j < B; j++){
			printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i*grid_size+j], i, g[i*grid_size+j] + dev_sums[i]);
		}
	}
		

	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);

	return 0;
}


__global__ void scan(float *g_odata, float *g_idata, int n) {
	/* CUDA Naive Scan Algorithm (double buffered). */

	extern __shared__ float temp[]; // allocated on invocation
	int thid = threadIdx.x;
	int pout = 0, pin = 1;

	// Load input into shared memory.
	// This is exclusive scan, so shift right by one
	// and set first element to 0
	temp[thid] = (thid > 0) ? g_idata[thid-1] : 0;
	__syncthreads();
	for (int offset = 1; offset < n; offset *= 2) {
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout*n+thid] += temp[pin*n+thid - offset];
		else
			temp[pout*n+thid] = temp[pin*n+thid];

		__syncthreads();
	}
	g_odata[thid] = temp[pout*n+thid]; // write output
}


__global__ void prescan(float *g_odata, float *g_idata, int n, float *g_sums) {
	/* CUDA Work-Efficient Scan Algorithm. */

	extern  __shared__  float temp[]; // allocated on invocation 
	int thid = threadIdx.x;  // thread id of a thread in a block
	int gthid = (blockIdx.x * blockDim.x) + thid; // global thread id of grid
	int offset = 1;

	// for each thread in a block, put data into shared memory
	if (gthid > n) {
		// handle non-power of two arrays by padding elements in last block
		temp[2*thid] = 0;
		temp[2*thid+1] = 0;
	}
	else {
		// grab data from input array
		temp[2*thid] = g_idata[2*gthid];
		temp[2*thid+1] = g_idata[2*gthid+1];
	}

    // build sum in place up the tree 
	for (int d = B>>1; d > 0; d >>= 1) { 
        __syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
		    	temp[bi] += temp[ai];         
  		}
  		offset *= 2; 
    } 

	if (thid == 0) { 
		g_sums[blockIdx.x] = temp[B - 1];
		temp[B - 1] = 0; 
	}

	// clear the last element 
	// traverse down tree & build scan
	for (int d = 1; d < B; d *= 2) { 
    	offset >>= 1; 
    	__syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			float t = temp[ai]; 
    		temp[ai] = temp[bi]; 
    		temp[bi] += t; 
    	} 
	} 
	__syncthreads(); 
	
	// write results to device memory 
	g_odata[2*gthid] = temp[2*thid]; 
	g_odata[2*gthid+1] = temp[2*thid+1]; 
}


void scanCPU(float *f_out, float *f_in, int i_n) {
	/* Apply all-prefix sums to an array on the CPu
	without parallelization. */

	f_out[0] = 0;

	/* for each array element, the value is the previous sum
	plus the current array value */
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
