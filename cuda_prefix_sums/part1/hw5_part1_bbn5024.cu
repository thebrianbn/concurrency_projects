// Author: Brian Nguyen


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 64 // number of array elements
#define B 4  // number of elements in a block

__global__ void scan(float *g_odata, float *g_idata, int n);
__global__ void prescan(float *g_odata, float *g_idata, int n, float *g_sums);
__global__ void uniform_add(float *o_array, float *sum_array);
void scanCPU(float *f_out, float *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end) {
	/* Calculate the time difference. */

	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
} 

int main() {
	/* Compare results between serial and parallel versions of the
	prefix-sums algorithm. */

	int grid_size = floor(N / B);
	int thread_size = B / 2;
	int grid_size2 = grid_size / B;

	// arrays to be used for initial, cpu-result, and gpu-result arrays
	// respectively.
	float a[N], c[N], g[N], sums[grid_size];
	timeval start, end;

	// temporary pointer arrays for computation
	float *dev_a, *dev_g, *dev_sums;
	int size = N * sizeof(float);
	int size_sums = grid_size * sizeof(float);
	int size_sums2 = grid_size2 * sizeof(float);

	double d_gpuTime, d_cpuTime;

	// initialize matrix a with random floats between 0 and 1000
	for (int i = 1; i <= N; i++) {
		a[i-1] = i;
	}

	// CPU version (serial) of prefix-sum
	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);

	// START OF FIRST PRE-SCAN RUN

	// initialize a and b matrices here for CUDA
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);
	hipMalloc((void **) &dev_sums, size_sums);

	// GPU version (CUDA) of prefix-sum
	gettimeofday(&start, NULL);
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	// work-efficient scan for SUMS array
	prescan<<<grid_size, thread_size, B*sizeof(float)>>>(dev_g, dev_a, N, dev_sums);
	hipDeviceSynchronize();
	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(sums, dev_sums, size_sums, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	
	hipFree(dev_a); hipFree(dev_g); hipFree(dev_sums);

	for (int j = 0; j < grid_size; j++) {
		printf("sums[%i] = %0.3f\n", j, sums[j]);
	}

	// START OF SECOND PRE-SCAN RUN

	float inc[grid_size], sums_inc[grid_size2], inc_final[grid_size2];
	float *dev_inc, *dev_sums_inc, *dev_inc_final, *dev_sums_input;

	hipMalloc((void **) &dev_sums_input, size_sums);
	hipMalloc((void **) &dev_inc, size_sums);
	hipMalloc((void **) &dev_sums_inc, size_sums2);

	hipMemcpy(dev_sums_input, sums, size_sums, hipMemcpyHostToDevice);

	prescan<<<grid_size2, thread_size, B*sizeof(float)>>>(dev_inc, dev_sums_input, grid_size2, dev_sums_inc);
	hipDeviceSynchronize();
	hipMemcpy(inc, dev_inc, size_sums, hipMemcpyDeviceToHost);
	hipMemcpy(sums_inc, dev_sums_inc, size_sums2, hipMemcpyDeviceToHost);

	hipFree(dev_inc); hipFree(dev_sums_inc); hipFree(dev_sums_input);

	
	for (int j = 0; j < grid_size2; j++) {
		printf("inc[%i] = %0.3f\n", j, sums_inc[j]);
	}
	for (int j = 0; j < grid_size; j++) {
		printf("inc[%i] = %0.3f\n", j, inc[j]);
	}

	scanCPU(inc_final, sums_inc, size_sums2);

	// START OF UPDATING SUMS

	for (int j = 0; j < grid_size2; j++) {
		printf("pscan2[%i] = %0.3f\n", j, inc_final[j]);
	}

	float g2[grid_size];
	float *dev_g2;

	hipMalloc((void **) &dev_g2, size);
	hipMalloc((void **) &dev_inc_final, size_sums);

	hipMemcpy(dev_inc_final, inc_final, size_sums, hipMemcpyHostToDevice);
	hipMemcpy(dev_g2, inc, size_sums, hipMemcpyHostToDevice);

	uniform_add<<<grid_size2, thread_size, B*sizeof(float)>>>(dev_g2, dev_inc_final);
	hipDeviceSynchronize();

	hipMemcpy(g2, dev_g2, size, hipMemcpyDeviceToHost);

	for (int j = 0; j < grid_size; j++) {
		printf("g2[%i] = %0.3f\n", j, g2[j]);
	}

	// START OF FINAL UPDATE TO FIRST PREFIX SCAN

	float g3[N], first_add[grid_size];
	float *dev_g3, *dev_first_add;

	hipMalloc((void **) &dev_g3, size);
	hipMalloc((void **) &dev_first_add, size_sums);

	hipMemcpy(dev_first_add, g2, size_sums, hipMemcpyHostToDevice);
	hipMemcpy(dev_g3, g, size, hipMemcpyHostToDevice);

	uniform_add<<<grid_size, thread_size, B*sizeof(float)>>>(dev_g3, dev_first_add);
	hipDeviceSynchronize();

	hipMemcpy(g3, dev_g3, size, hipMemcpyDeviceToHost);

	// display results of the prefix-sum
	for (int i = 0; i < N; i++) {
		printf("c[%i] = %0.3f, g3[%i] = %0.3f\n", i, c[i], i, g3[i]);
		//if (c[i] != g[i])
		//{
		//	printf("Results do not match! c[%i]=%f, g[%i]=%f\n", i, c[i], i, g[i]);
		//	break;
		//}
	}
		
	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);

	return 0;
}


__global__ void scan(float *g_odata, float *g_idata, int n) {
	/* CUDA Naive Scan Algorithm (double buffered). */

	extern __shared__ float temp[]; // allocated on invocation
	int thid = threadIdx.x;
	int pout = 0, pin = 1;

	// Load input into shared memory.
	// This is exclusive scan, so shift right by one
	// and set first element to 0
	temp[thid] = (thid > 0) ? g_idata[thid-1] : 0;
	__syncthreads();
	for (int offset = 1; offset < n; offset *= 2) {
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout*n+thid] += temp[pin*n+thid - offset];
		else
			temp[pout*n+thid] = temp[pin*n+thid];

		__syncthreads();
	}
	g_odata[thid] = temp[pout*n+thid]; // write output
}


__global__ void prescan(float *g_odata, float *g_idata, int n, float *g_sums) {
	/* CUDA Work-Efficient Scan Algorithm. */

	extern  __shared__  float temp[]; // allocated on invocation 
	int thid = threadIdx.x;  // thread id of a thread in a block
	int gthid = (blockIdx.x * blockDim.x) + thid; // global thread id of grid
	int offset = 1;

	/*
	// for each thread in a block, put data into shared memory
	if (gthid > n) {
		// handle non-power of two arrays by padding elements in last block
		temp[2*thid] = 0;
		temp[2*thid+1] = 0;
	}
	else {
		// grab data from input array
		temp[2*thid] = g_idata[2*gthid];
		temp[2*thid+1] = g_idata[2*gthid+1];
	}
	*/
	temp[2*thid] = g_idata[2*gthid];
	temp[2*thid+1] = g_idata[2*gthid+1];

    // build sum in place up the tree 
	for (int d = B>>1; d > 0; d >>= 1) { 
        __syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
		    	temp[bi] += temp[ai];         
  		}
  		offset *= 2; 
    } 

	if (thid == 0) { 
		g_sums[blockIdx.x] = temp[B - 1];
		temp[B - 1] = 0; 
	}

	// clear the last element 
	// traverse down tree & build scan
	for (int d = 1; d < B; d *= 2) { 
    	offset >>= 1; 
    	__syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			float t = temp[ai]; 
    		temp[ai] = temp[bi]; 
    		temp[bi] += t; 
    	} 
	} 
	__syncthreads(); 
	
	// write results to device memory 
	g_odata[2*gthid] = temp[2*thid]; 
	g_odata[2*gthid+1] = temp[2*thid+1]; 
}

__global__ void uniform_add(float *o_array, float *sum_array) {

	int bid = blockIdx.x;
	int gthid = (bid * blockDim.x) + threadIdx.x; // global thread id of grid

	o_array[2*gthid] = o_array[2*gthid] + sum_array[bid];
	o_array[2*gthid+1] = o_array[2*gthid+1] + sum_array[bid];
}


void scanCPU(float *f_out, float *f_in, int i_n) {
	/* Apply all-prefix sums to an array on the CPu
	without parallelization. */

	f_out[0] = 0;

	/* for each array element, the value is the previous sum
	plus the current array value */
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
