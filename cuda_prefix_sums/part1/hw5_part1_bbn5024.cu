// Author: Brian Nguyen


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define N 1024 // number of array elements
#define B 4  // number of elements in a block

__global__ void scan(double *g_odata, double *g_idata, int n);
__global__ void prescan(double *g_odata, double *g_idata, int n, double *g_sums);
__global__ void uniform_add(double *o_array, double *sum_array);
void scanCPU(double *f_out, double *f_in, int i_n);

bool isPowerTwo(ulong x) {
    return (x & (x - 1)) == 0;
}

double myDiffTime(struct timeval &start, struct timeval &end) {
	/* Calculate the time difference. */

	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
} 

int main() {
	/* Compare results between serial and parallel versions of the
	prefix-sums algorithm. */

	int grid_size = ceil(N / B);  // size of grids for first prefix-scan
	int grid_size2 = ceil(grid_size / B); // size of grids for second prefix-scan
	int thread_size = B / 2;  // thread size for each block

	// arrays to be used for initial, cpu-result, and gpu-result arrays
	// respectively.
	double a[N], c[N], g[N], sums[grid_size];
	timeval start, end;

	// temporary pointer arrays for computation
	double *dev_a, *dev_g, *dev_sums;
	int size = N * sizeof(double);
	int size_sums = grid_size * sizeof(double);
	int size_sums2 = grid_size2 * sizeof(double);

	double d_gpuTime, d_cpuTime;

	// initialize matrix a with random doubles between 0 and 1000
	for (int i = 0; i <= N; i++) {
		a[i] = (double)(rand() % 1000000) / 1000.0;
	}
	/*
	if (!isPowerTwo(N)) {
		next_power = pow(2, ceil(log(x)/log(2)));
	}
	*/

	// CPU version (serial) of prefix-sum
	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);

	// START OF FIRST PRE-SCAN RUN

	// initialize a and b matrices here for CUDA
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);
	hipMalloc((void **) &dev_sums, size_sums);

	// GPU version (CUDA) of prefix-sum
	gettimeofday(&start, NULL);
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	// work-efficient scan for SUMS array
	prescan<<<grid_size, thread_size, B*sizeof(double)>>>(dev_g, dev_a, N, dev_sums);
	hipDeviceSynchronize();
	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(sums, dev_sums, size_sums, hipMemcpyDeviceToHost);
	
	hipFree(dev_a); hipFree(dev_g); hipFree(dev_sums);

	// START OF SECOND PRE-SCAN RUN

	double inc[grid_size], sums_inc[grid_size2], inc_final[grid_size2];
	double *dev_inc, *dev_sums_inc, *dev_inc_final, *dev_sums_input;

	hipMalloc((void **) &dev_sums_input, size_sums);
	hipMalloc((void **) &dev_inc, size_sums);
	hipMalloc((void **) &dev_sums_inc, size_sums2);

	hipMemcpy(dev_sums_input, sums, size_sums, hipMemcpyHostToDevice);

	prescan<<<grid_size2, thread_size, B*sizeof(double)>>>(dev_inc, dev_sums_input, grid_size2, dev_sums_inc);
	hipDeviceSynchronize();
	hipMemcpy(inc, dev_inc, size_sums, hipMemcpyDeviceToHost);
	hipMemcpy(sums_inc, dev_sums_inc, size_sums2, hipMemcpyDeviceToHost);

	hipFree(dev_inc); hipFree(dev_sums_inc); hipFree(dev_sums_input);

	scanCPU(inc_final, sums_inc, size_sums2);

	// START OF UPDATING SUMS

	double g2[grid_size];
	double *dev_g2;

	hipMalloc((void **) &dev_g2, size);
	hipMalloc((void **) &dev_inc_final, size_sums);

	hipMemcpy(dev_inc_final, inc_final, size_sums, hipMemcpyHostToDevice);
	hipMemcpy(dev_g2, inc, size_sums, hipMemcpyHostToDevice);

	uniform_add<<<grid_size2, thread_size, B*sizeof(double)>>>(dev_g2, dev_inc_final);
	hipDeviceSynchronize();

	hipMemcpy(g2, dev_g2, size, hipMemcpyDeviceToHost);

	// START OF FINAL UPDATE TO FIRST PREFIX SCAN

	double g3[N];
	double *dev_g3, *dev_first_add;

	hipMalloc((void **) &dev_g3, size);
	hipMalloc((void **) &dev_first_add, size_sums);

	hipMemcpy(dev_first_add, g2, size_sums, hipMemcpyHostToDevice);
	hipMemcpy(dev_g3, g, size, hipMemcpyHostToDevice);

	uniform_add<<<grid_size, thread_size, B*sizeof(double)>>>(dev_g3, dev_first_add);
	hipDeviceSynchronize();

	hipMemcpy(g3, dev_g3, size, hipMemcpyDeviceToHost);

	gettimeofday(&end, NULL);
	d_gpuTime = myDiffTime(start, end);

	hipFree(dev_g3); hipFree(dev_first_add);

	// display results of the prefix-sum
	for (int i = 0; i < N; i++) {
		printf("c[%i] = %0.3f, g3[%i] = %0.3f\n", i, c[i], i, g3[i]);
		//if (c[i] != g[i])
		//{
		//	printf("Results do not match! c[%i]=%f, g[%i]=%f\n", i, c[i], i, g[i]);
		//	break;
		//}
	}
		
	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);

	return 0;
}


__global__ void scan(double *g_odata, double *g_idata, int n) {
	/* CUDA Naive Scan Algorithm (double buffered). */

	extern __shared__ double temp[]; // allocated on invocation
	int thid = threadIdx.x;
	int pout = 0, pin = 1;

	// Load input into shared memory.
	// This is exclusive scan, so shift right by one
	// and set first element to 0
	temp[thid] = (thid > 0) ? g_idata[thid-1] : 0;
	__syncthreads();
	for (int offset = 1; offset < n; offset *= 2) {
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout*n+thid] += temp[pin*n+thid - offset];
		else
			temp[pout*n+thid] = temp[pin*n+thid];

		__syncthreads();
	}
	g_odata[thid] = temp[pout*n+thid]; // write output
}


__global__ void prescan(double *g_odata, double *g_idata, int n, double *g_sums) {
	/* CUDA Work-Efficient Scan Algorithm. */

	extern  __shared__  double temp[]; // allocated on invocation 
	int thid = threadIdx.x;  // thread id of a thread in a block
	int gthid = (blockIdx.x * blockDim.x) + thid; // global thread id of grid
	int offset = 1;

	/*
	// for each thread in a block, put data into shared memory
	if (gthid > n) {
		// handle non-power of two arrays by padding elements in last block
		temp[2*thid] = 0;
		temp[2*thid+1] = 0;
	}
	else {
		// grab data from input array
		temp[2*thid] = g_idata[2*gthid];
		temp[2*thid+1] = g_idata[2*gthid+1];
	}
	*/
	temp[2*thid] = g_idata[2*gthid];
	temp[2*thid+1] = g_idata[2*gthid+1];

    // build sum in place up the tree 
	for (int d = B>>1; d > 0; d >>= 1) { 
        __syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
		    	temp[bi] += temp[ai];         
  		}
  		offset *= 2; 
    } 

	if (thid == 0) { 
		g_sums[blockIdx.x] = temp[B - 1];
		temp[B - 1] = 0; 
	}

	// clear the last element 
	// traverse down tree & build scan
	for (int d = 1; d < B; d *= 2) { 
    	offset >>= 1; 
    	__syncthreads(); 
		if (thid < d) { 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			double t = temp[ai]; 
    		temp[ai] = temp[bi]; 
    		temp[bi] += t; 
    	} 
	} 
	__syncthreads(); 
	
	// write results to device memory 
	g_odata[2*gthid] = temp[2*thid]; 
	g_odata[2*gthid+1] = temp[2*thid+1]; 
}

__global__ void uniform_add(double *o_array, double *sum_array) {

	int bid = blockIdx.x;
	int gthid = (bid * blockDim.x) + threadIdx.x; // global thread id of grid

	o_array[2*gthid] = o_array[2*gthid] + sum_array[bid];
	o_array[2*gthid+1] = o_array[2*gthid+1] + sum_array[bid];
}


void scanCPU(double *f_out, double *f_in, int i_n) {
	/* Apply all-prefix sums to an array on the CPu
	without parallelization. */

	f_out[0] = 0;

	/* for each array element, the value is the previous sum
	plus the current array value */
	for (int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
